#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "rdma_common.h"

struct ibv_mr *mr = NULL;
int totalsize = TOTALSIZE;
int buffsize = BUFFSIZE;
void *addr, *addrserver;

/**
 * @brief All of the functions with an address is run from the transmitter, the rest is ran from the receiver and uses the global *addr.
 * 
 */



/* This function registers RDMA memory region on GPU */
extern "C" struct ibv_mr* rdma_gpubuffer_alloc(struct ibv_pd *pd, uint32_t length,
    enum ibv_access_flags permission)
{
	if (!pd) {
		rdma_error("Protection domain is NULL \n");
		return NULL;
	}

	hipMalloc((void**)&addr, length);
	if (!addr) {
		rdma_error("failed to allocate buffer, -ENOMEM\n");
		return NULL;
	}
	printf("Allocating gpu memory\n");
	debug("GPU Buffer allocated: %p , len: %u \n", addr, length);

	if (!pd) {
		rdma_error("Protection domain is NULL, ignoring \n");
		return NULL;
	}
	
	debug("GPU pointer address : %p\n", addr);
	mr = ibv_reg_mr(pd, addr, length, permission);
	debug("mr: %p ", mr);
	if (!mr) {
		rdma_error("Failed to create mr on buffer, errno: %s \n", strerror(errno));
		hipFree(addr);
	}

	debug("Registered: %p , len: %u , stag: 0x%x \n",
	      mr->addr,
	      (unsigned int) mr->length,
	      mr->lkey);

	return mr;
}

extern "C" struct ibv_mr* rdma_gpubuffer_alloc_adress(struct ibv_pd *pd, void* addr, uint32_t length,
    enum ibv_access_flags permission)
{
	if (!pd) {
		rdma_error("Protection domain is NULL \n");
		return NULL;
	}
	//debug("GPU Buffer from server: %p , len: %u \n", addr, length);
	mr = ibv_reg_mr(pd, addr, length, permission);
	debug("mr: %p ", mr);
	if (!mr) {
		rdma_error("Failed to create mr on buffers, errno: %s \n", strerror(errno));
		hipFree(addr);
	}

	debug("Registered: %p , len: %u , stag: 0x%x \n",
	      mr->addr,
	      (unsigned int) mr->length,
	      mr->lkey);



	return mr;
}

extern "C" int cuAlloc(void** addr, size_t length){
	void *cudmem; 
	hipError_t err = hipHostMalloc((void**)&cudmem, length);
	if (err != hipSuccess) {
		rdma_error("Failed to allocate gpumemory, -ENOMEM\n");
		return err;
	}
	debug("GPU Buffer from server: %p , len: %lu \n", cudmem, (unsigned long)length);

	*addr = cudmem;
	return (int)err;

}

extern "C" int cuFree(void* addr){

	hipError_t err = hipFree(addr);
	if (err != hipSuccess) {
		rdma_error("Failed to allocate gpumemory, -ENOMEM\n");
		return -1;
	}
	//printf("Cuda Error? :%d",err);
	return (int)err;
}


/* This function releases RDMA memory region on GPU */
extern "C" void rdma_gpubuffer_free()
{
        if (!mr) {
	       rdma_error("Passed memory region is NULL, ignoring\n");
		return ;
	}
	void *to_free = mr->addr;

	debug("Deregistered: %p , len: %u , stag : 0x%x \n",
	      mr->addr,
	      (unsigned int) mr->length,
	      mr->lkey);
	ibv_dereg_mr(mr);

	debug("Buffer %p free'ed\n", to_free);
	hipFree(to_free);
}

extern "C" void rdma_gpubuffer_free_addr(struct ibv_mr* mr)
{
        if (!mr) {
	       rdma_error("Passed memory region is NULL, ignoring\n");
		return ;
	}
	void *to_free = mr->addr;

	debug("Deregistered: %p , len: %u , stag : 0x%x \n",
	      mr->addr,
	      (unsigned int) mr->length,
	      mr->lkey);
	ibv_dereg_mr(mr);

	debug("Buffer %p free'ed\n", to_free);
	hipFree(to_free);
}
/* 
	Function reads the value of the first address. If the memory is equal to idx we know that the transmitter has written the first sequence.
	We then write a -1 in that memory location to signal we've read the written memory.
	If the idx reaches the amount of packages(idx) specified, we break.
*/
__global__ void kernel(void *addr, int totalsize, int buffsize)
{
	volatile int* memory = (int*)addr;
	int idx = 1;
	printf("Clientside GPU \n");
	while(1){		
		if(memory[0] == idx){
			printf("Read full buffer of msg #: %d\n", memory[0]);
				
				//Print out entire written memory buffer
				//for (size_t i = 0; i < buffsize/sizeof(int); i++)
				//{
					//printf(" %d", (memory[i]));
				//}
				
			memory[0] = -1;
			idx++;
		}
		if(idx == 20000)
		{
			memory[0] = -1;
			printf("Finished transfer, sent %d packages.\n", idx);
			break;
		}
	}		
}


__global__ void kernel_addr(void *addr, int totalsize, int buffsize)
{
	int* mem = (int*)addr;
	printf("Serverside GPU Array:");
	for (size_t i = 0; i < buffsize/4; i++)
	{
		printf(" %d", (mem[i]));
	}
	printf("\n");
}

extern "C" void kernel_start()
{
	kernel<<<1, 1>>>(addr, totalsize, buffsize); 
}

extern "C" int cuCopy(void* dst, void* src, size_t size)
{
	hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		rdma_error("Failed Memcpy: %d\n", err);
	}
	return err;
}

/**
 * @brief Copy a buffer to GPU memory.
 * 
 */
extern "C" void kernel_start_addr(void *addr)
{
	int test[buffsize] = {0};
	for (int i = 0; i < buffsize; i++)
	{
		test[i] = i;
	}
	debug("buffsize: %d\n",buffsize);
	hipError_t err = hipMemcpy(addr, (void*)&test, buffsize, hipMemcpyDefault);
	if (err != hipSuccess) {
		rdma_error("Failed Memcpy\n");
	}

}